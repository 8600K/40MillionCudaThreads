#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


/*
This was a learning project in which I was using thread blocks 
and a very large number of threads (40 Million) to compute atomically.
*/



using namespace std;


__global__ void divide(int *buff)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int i = index % 10;
	
	atomicAdd(&buff[i], i);

}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	const unsigned int ThreadNum = 40000000;
	const unsigned int BlockWidth = 1000;
	
	const int ArraySize = 10;
	const int ArrayBytes = ArraySize * sizeof(int);

	int hostArray[ArraySize];

	int * deviceArray;
	hipMalloc((void **)&deviceArray, ArrayBytes);
	hipMemset((void *)deviceArray, 0, ArrayBytes);
	hipEventRecord(start);
	//ThreadNum / BlockWidth = 40,000 Thread Blocks.  BlockWidth = 1000, which means 1000 Threads for each Thread Block, 40,000 x 1,000 = 40,000,000;
	divide << <ThreadNum / BlockWidth, BlockWidth >> > (deviceArray);
	hipEventRecord(stop);
	hipMemcpy(hostArray, deviceArray, ArrayBytes, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < 10; i++) {
		cout << hostArray[i] << endl;
	}

	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	cout <<"Time: " << ms << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(deviceArray);
	return 0;

}
